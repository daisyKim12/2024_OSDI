#include "hip/hip_runtime.h"
#include <pybind11/pybind11.h>
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <cstdint>
#include <iostream>


namespace py = pybind11;

__global__ void search_kernel 
(
  int* graph,
  int* queries,
  int* results,
  int topk
)
{

  const auto query_id = blockIdx.y;
  extern __shared__ std::uint32_t smem[];

  // Set smem working buffer for the distance calculation
  dataset_desc.set_smem_ptr(distance_work_buffer_ptr);

  // A flag for filtering.
  auto filter_flag = terminate_flag;

  const DATA_T* const query_ptr = queries_ptr + query_id * dataset_desc.dim;
  dataset_desc.template copy_query<DATASET_BLOCK_DIM>(
    query_ptr, query_buffer, query_smem_buffer_length);

  if (threadIdx.x == 0) {
    terminate_flag[0] = 0;
    topk_ws[0]        = ~0u;
  }

  // init hashmap need hashmap
  //
  //

  // compute distance to randomly selecting nodes
  const INDEX_T* const local_seed_ptr = seed_ptr ? seed_ptr + (num_seeds * query_id) : nullptr;
  device::compute_distance_to_random_nodes<TEAM_SIZE, DATASET_BLOCK_DIM>(result_indices_buffer,
                                                                         result_distances_buffer,
                                                                         query_buffer,
                                                                         dataset_desc,
                                                                         result_buffer_size,
                                                                         num_distilation,
                                                                         rand_xor_mask,
                                                                         local_seed_ptr,
                                                                         num_seeds,
                                                                         local_visited_hashmap_ptr,
                                                                         hash_bitlen,
                                                                         metric);
  __syncthreads();

  std::uint32_t iter = 0;
  while (1) {
    // sort
    if constexpr (TOPK_BY_BITONIC_SORT) {

      const unsigned multi_warps_1 = ((blockDim.x >= 64) && (MAX_CANDIDATES > 128)) ? 1 : 0;
      const unsigned multi_warps_2 = ((blockDim.x >= 64) && (MAX_ITOPK > 256)) ? 1 : 0;

      // reset small-hash table.
      if ((iter + 1) % small_hash_reset_interval == 0) {

        unsigned hash_start_tid;
        if (blockDim.x == 32) {
          hash_start_tid = 0;
        } else if (blockDim.x == 64) {
          if (multi_warps_1 || multi_warps_2) {
            hash_start_tid = 0;
          } else {
            hash_start_tid = 32;
          }
        } else {
          if (multi_warps_1 || multi_warps_2) {
            hash_start_tid = 64;
          } else {
            hash_start_tid = 32;
          }
        }
        hashmap::init(local_visited_hashmap_ptr, hash_bitlen, hash_start_tid);
      }

      // topk with bitonic sort
      if (std::is_same<SAMPLE_FILTER_T,
                       raft::neighbors::filtering::none_cagra_sample_filter>::value ||
          *filter_flag == 0) {
        topk_by_bitonic_sort<MAX_ITOPK, MAX_CANDIDATES>(result_distances_buffer,
                                                        result_indices_buffer,
                                                        internal_topk,
                                                        result_distances_buffer + internal_topk,
                                                        result_indices_buffer + internal_topk,
                                                        search_width * graph_degree,
                                                        topk_ws,
                                                        (iter == 0),
                                                        multi_warps_1,
                                                        multi_warps_2);
        __syncthreads();
      } else {
        topk_by_bitonic_sort_1st<MAX_ITOPK + MAX_CANDIDATES>(
          result_distances_buffer,
          result_indices_buffer,
          internal_topk + search_width * graph_degree,
          internal_topk,
          false);
        if (threadIdx.x == 0) { *terminate_flag = 0; }
      }
    } else {
      // topk with radix block sort
      topk_by_radix_sort<MAX_ITOPK, INDEX_T>{}(
        internal_topk,
        gridDim.x,
        result_buffer_size,
        reinterpret_cast<std::uint32_t*>(result_distances_buffer),
        result_indices_buffer,
        reinterpret_cast<std::uint32_t*>(result_distances_buffer),
        result_indices_buffer,
        nullptr,
        topk_ws,
        true,
        reinterpret_cast<std::uint32_t*>(smem_work_ptr));

      // reset small-hash table
      if ((iter + 1) % small_hash_reset_interval == 0) {
        hashmap::init(local_visited_hashmap_ptr, hash_bitlen);
      }
    }
    __syncthreads();

    if (iter + 1 == max_iteration) { break; }

    // pick up next parents
    if (threadIdx.x < 32) {
      pickup_next_parents<TOPK_BY_BITONIC_SORT, INDEX_T>(terminate_flag,
                                                         parent_list_buffer,
                                                         result_indices_buffer,
                                                         internal_topk,
                                                         dataset_desc.size,
                                                         search_width);
    }

    if (*terminate_flag && iter >= min_iteration) { break; }

    // compute the norms between child nodes and query node
    constexpr unsigned max_n_frags = 8;
    device::compute_distance_to_child_nodes<TEAM_SIZE, DATASET_BLOCK_DIM, max_n_frags>(
      result_indices_buffer + internal_topk,
      result_distances_buffer + internal_topk,
      query_buffer,
      dataset_desc,
      knn_graph,
      graph_degree,
      local_visited_hashmap_ptr,
      hash_bitlen,
      parent_list_buffer,
      result_indices_buffer,
      search_width,
      metric);
    __syncthreads();

    // filtering
    //
    //

    iter++;
  }

  for (std::uint32_t i = threadIdx.x; i < top_k; i += blockDim.x) {
    unsigned j  = i + (top_k * query_id);
    unsigned ii = i;
    if (TOPK_BY_BITONIC_SORT) { ii = device::swizzling(i); }
    if (result_distances_ptr != nullptr) { result_distances_ptr[j] = result_distances_buffer[ii]; }
    constexpr INDEX_T index_msb_1_mask = utils::gen_index_msb_1_mask<INDEX_T>::value;

    result_indices_ptr[j] =
      result_indices_buffer[ii] & ~index_msb_1_mask;  // clear most significant bit
  }
  if (threadIdx.x == 0 && num_executed_iterations != nullptr) {
    num_executed_iterations[query_id] = iter + 1;
  }
}


torch::Tensor search(torch::Tensor graph, torch::Tensor queries) {
  
  int block_size = 64;
  int num_queries = queries.size(0);
  int topk = 10;

  torch::Tensor results = torch::zeros({num_queries, topk}, torch::device(queries.device()).dtype(torch::kInt32));
  
  dim3 thread_dims(block_size, 1, 1);
  dim3 block_dims(1, num_queries, 1);

  search_kernel<<<block_dims, thread_dims>>>(
    graph.data_ptr<int>(),
    queries.data_ptr<int>(),
    results.data_ptr<int>(),
    topk
  );  

  return results;

}

PYBIND11_MODULE(TORCH_EXTENSION_NAME, m) {
    m.def("search", &search, "A function that performs search on the graph");
}